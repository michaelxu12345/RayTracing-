#include "hip/hip_runtime.h"
#include "common.cuh"
#include "SDL.h"
#include "processing.cu"
#include "hittable.cuh"
#include "hittable_list.cuh"
#include "sphere.cuh"
#include <hiprand/hiprand_kernel.h>


__global__ void create_world(hittable** d_list, hittable** d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_list[0] = new sphere(vec3(0, 0, -1), 0.5,
                        new lambertian(vec3(0.1, 0.1, 0.8)));
        d_list[1] = new sphere(vec3(0, -100.5, -1), 100.0,
                        new lambertian(vec3(0.1, 0.5, 0.2)));
        d_list[2] = new sphere(vec3(-1, 0, -1), 0.5,
                        new dielectric(1.5));
        d_list[3] = new sphere(vec3(-1.0, 0.0, -1), -0.4,
                        new dielectric(1.5));
        d_list[4] = new sphere(vec3(1.0, 0.0, -1), 0.5,
                        new metal(vec3(0.8, 0.6, 0.2), 0.0));
        *d_world = new hittable_list(d_list, 5);


        /*d_list[0] = new sphere(vec3(0, 0, -1), 0.5,
                        new metal(vec3(0.8, 0.2, 0.1), 0.0));
        d_list[1] = new sphere(vec3(0, -100.5, -1), 0.5,
                        new lambertian(vec3(0.1, 0.2, 0.5)));
        *d_world = new hittable_list(d_list, 2);*/
    }
}

__global__ void render_init(int width, int height, hiprandState* rand_state) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    /*if ((x >= width) || (y >= height)) {
        return;
    }*/

    int pixel_index = y * width + x;
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);

    //int idx = (y * width + x) * 3;
}

int main(int argc, char* argv[]) {

    // Initialize SDL
    if (SDL_Init(SDL_INIT_VIDEO) < 0) {
        std::cerr << "Failed to initialize SDL: " << SDL_GetError() << std::endl;
        return -1;
    }

    SDL_Window* window = SDL_CreateWindow("CUDA Image Renderer", SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED, 1280, 720, SDL_WINDOW_SHOWN);
    if (!window) {
        std::cerr << "Failed to create window: " << SDL_GetError() << std::endl;
        SDL_Quit();
        return -1;
    }

    SDL_Renderer* renderer = SDL_CreateRenderer(window, -1, SDL_RENDERER_ACCELERATED);
    if (!renderer) {
        std::cerr << "Failed to create renderer: " << SDL_GetError() << std::endl;
        SDL_DestroyWindow(window);
        SDL_Quit();
        return -1;
    }

    // Create an SDL texture
    int width = 1280;
    int height = 720;
    SDL_Texture* texture = SDL_CreateTexture(renderer, SDL_PIXELFORMAT_RGB24, SDL_TEXTUREACCESS_STREAMING, width, height);

    // Allocate memory for image
    unsigned char* image;
    hipMallocManaged((void**)&image, height * width * 3);
        
    // new unsigned char[width * height * 3]; // Assuming RGB format

    // Setting up cam
    camera cam;

    cam.aspect_ratio = float(width) / float(height);
    cam.image_width = width;
    cam.image_height = height;
    cam.vfov = 90;
    cam.lookfrom = point3(-2, 2, 1);
    cam.lookat = point3(0, 1, -1);
    cam.vup = vec3(0, 1, 0);
    cam.num_samples = 1;

    cam.initialize();

    // Setting random numbers
    hiprandState* d_rand_state;
    hipMalloc((void**)&d_rand_state, width * height * sizeof(hiprandState));

    dim3 blockSize(16, 16);
    dim3 gridSize((cam.image_width + blockSize.x - 1) / blockSize.x, (cam.image_height + blockSize.y - 1) / blockSize.y);

    render_init << <gridSize, blockSize >> > (width, height, d_rand_state);
    checkError(__FILE__, __LINE__);
    hipDeviceSynchronize();

    // Creating world
    hittable** d_list;
    hittable** d_world;
    hipMalloc((void**)&d_list, 5 * sizeof(hittable*));
    hipMalloc((void**)&d_world, sizeof(hittable*));

    create_world <<<1, 1 >>> (d_list, d_world);
    checkError(__FILE__, __LINE__);
    hipDeviceSynchronize();

    

    processImage(image, d_world, cam, d_rand_state);

    //hipDeviceSynchronize();
    
    std::cout << "processed image" << std::endl;

    // Update texture with the processed image
    SDL_UpdateTexture(texture, NULL, image, width * 3);

    std::cout << "updated texture" << std::endl;
    // Main loop
    bool running = true;
    SDL_Event event;
    while (running) {
        while (SDL_PollEvent(&event)) {
            if (event.type == SDL_QUIT) {
                running = false;
            }
            else if (event.type == SDL_KEYDOWN) {
                switch (event.key.keysym.sym) {
                case SDLK_w:
                    // Move camera forward
                    cam.lookfrom += vec3(0, 0, -0.1);
                    cam.lookat += vec3(0, 0, -0.1);
                    break;
                case SDLK_s:
                    // Move camera backward
                    cam.lookfrom += vec3(0, 0, 0.1);
                    cam.lookat += vec3(0, 0, 0.1);
                    break;
                case SDLK_a:
                    // Move camera left
                    cam.lookfrom += vec3(-0.1, 0, 0);
                    cam.lookat += vec3(-0.1, 0, 0);
                    break;
                case SDLK_d:
                    // Move camera right
                    cam.lookfrom += vec3(0.1, 0, 0);
                    cam.lookat += vec3(0.1, 0, 0);
                    break;
                case SDLK_UP:
                    // Look up (rotate around the x-axis)
                    cam.lookat += vec3(0, 0.1, 0);
                    break;
                case SDLK_DOWN:
                    // Look down (rotate around the x-axis)
                    cam.lookat += vec3(0, -0.1, 0);
                    break;
                case SDLK_LEFT:
                    // Look left (rotate around the y-axis)
                    cam.lookat += vec3(-0.1, 0, 0);
                    break;
                case SDLK_RIGHT:
                    // Look right (rotate around the y-axis)
                    cam.lookat += vec3(0.1, 0, 0);
                    break;
                }

                // Reinitialize camera and reprocess image after moving
                cam.initialize();
                processImage(image, d_world, cam, d_rand_state);
                SDL_UpdateTexture(texture, NULL, image, width * 3);
            }
        }

        SDL_RenderClear(renderer);

        // Render the image
        SDL_RenderCopy(renderer, texture, NULL, NULL);

        // Render the button
        //SDL_SetRenderDrawColor(renderer, 0, 0, 255, 255); // Blue button
        //SDL_RenderFillRect(renderer, &buttonRectRight);

        //SDL_SetRenderDrawColor(renderer, 255, 0, 0, 255); // Blue button
        //SDL_RenderFillRect(renderer, &buttonRectLeft);

        // Optionally, render button text (requires SDL_ttf for text rendering, not covered here)

        SDL_RenderPresent(renderer);
    }

    // Clean up
    delete[] image;
    SDL_DestroyTexture(texture);
    SDL_DestroyRenderer(renderer);
    SDL_DestroyWindow(window);
    SDL_Quit();

    return 0;
}